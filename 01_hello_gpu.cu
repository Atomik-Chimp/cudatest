
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 * <<Done>>
 */

__global__ void helloGPU()
{
  printf("Hello from the GPU. Checking that this code is executed...\n");
}

int main()
{

  helloCPU();

  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   * <<Done>>
   */

  helloGPU<<<1,1>>>();

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
   
   hipDeviceSynchronize();
}
