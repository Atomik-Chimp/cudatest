
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;
  hipError_t err;

  size_t size = N * sizeof(int);
  err = hipMallocManaged(&a, size);
  
  if(err != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(err));
  }
  
  init(a, N);

  /*
   * Error checking reveals that the thread size was out of range
   * threads are limited to 1024 or less
   *
   */
  
  /* size_t threads_per_block = 2048; */
  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;
  
  /* 
   * The example code used two error variables, one before sync and one after.
   * There were no sync errors though, just the one above
   */

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  err = hipGetLastError();
  if(err != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(err));
  }
  
  err = hipDeviceSynchronize();
  if(err != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(err));
  }

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
